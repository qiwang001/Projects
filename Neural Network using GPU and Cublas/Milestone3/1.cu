#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 5  // Number of rows in A and C
#define N 4  // Number of columns in B and C
#define K 3  // Number of columns in A and rows in B

void initialize_W_and_B_on_GPU(double* W_h[], double* B_h[], double* W[], double*B[],int L, int nhidden)
{
W[0]=NULL;
B[0]=NULL;
hipMalloc((void **) &W[1], SIZE* nhidden * sizeof(double));
hipMalloc((void **) &B[1], SIZE* nhidden * sizeof(double));

double stddev = sqrt(2.0/SIZE);
for(int i=0;i<SIZE*nhidden;i++)
  W[1][i] = stddev * ((double) rand() / (RAND_MAX / 2.0f) - 1.0f);

stddev = sqrt(2.0/nhidden);
for(int i=0;i<nhidden;i++)
  B[1][i] = stddev * ((double) rand() / (RAND_MAX / 2.0f) - 1.0f);

for(int i=2;i <L-1 ;i++)
{
  hipMalloc((void **) &W[i], nhidden* nhidden * sizeof(double));
  for(int j=0;j<nhidden*nhidden;j++)
      W[i][j] = stddev * ((double) rand() / (RAND_MAX / 2.0f) - 1.0f);
  
  hipMalloc((void **) &B[i], nhidden * sizeof(double));
  for(int j=0;j<nhidden;j++)
      B[i][j] = stddev * ((double) rand() / (RAND_MAX / 2.0f) - 1.0f);

}
hipMalloc((void **) &W[L-1], OUT_NUM* nhidden * sizeof(double));
  for(int j=0;j<OUT_NUM*nhidden;j++)
      W[L-1][j] = stddev * ((double) rand() / (RAND_MAX / 2.0f) - 1.0f);

hipMalloc((void **) &B[L-1], OUT_NUM * sizeof(double));
stddev = sqrt(2.0/OUT_NUM);
for(int j=0;j<OUT_NUM;j++)
    B[L-1][j] = stddev * ((double) rand() / (RAND_MAX / 2.0f) - 1.0f);
}


int
main()
{
hipblasHandle_t handle;
hipblasCreate(&handle);
double * a_h = (double*) malloc(M*K * sizeof(double));
double * b_h = (double*) malloc(K*N * sizeof(double));
double * c_h = (double*) malloc(M*N * sizeof(double));
for(int i=0;i<M*K;i++)
    a_h[i] = i+1;

for(int i=0;i<K*N;i++)
    b_h[i] = i+1;

for(int i=0;i<M*N;i++)
    c_h[i] = 101+1;

double * a;
hipMalloc((void**) &a,M*K * sizeof(double));
double * b;
hipMalloc((void**) &b,K*N * sizeof(double));
double * c;
hipMalloc((void**) &c,M*N * sizeof(double));

hipMemcpy(a, a_h,M*K*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(b, b_h,K*N*sizeof(double), hipMemcpyHostToDevice);
double alpha = 1;
double beta = 0;
hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,M, N, K,&alpha, a, M, b, K,& beta,c,M);
beta = 0.1;
hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,M, N, K,&alpha, a, M, b, K,& beta,c,M);
hipMemcpy(c_h, c,M*N*sizeof(double), hipMemcpyDeviceToHost);
for(int i=0;i<8;i++)
    printf("%f ", c_h[i]);
hipblasDestroy(handle);
return 0;
}
